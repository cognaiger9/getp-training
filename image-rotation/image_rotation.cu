#include "hip/hip_runtime.h"
#include <cstdio>

#include "image_rotation.h"

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

// Device(GPU) pointers
static float *input_images_gpu, *output_images_gpu;

void rotate_image_native(float *input_images, float *output_images, int W, int H,
                        float sin_theta, float cos_theta, int num_src_images) {
  float x0 = W / 2.0f;
  float y0 = H / 2.0f;

  // Rotate images
  for (int i = 0; i < num_src_images; i++) {
    for (int dest_x = 0; dest_x < W; dest_x++) {
      for (int dest_y = 0; dest_y < H; dest_y++) {
        float xOff = dest_x - x0;
        float yOff = dest_y - y0;
        int src_x = (int) (xOff * cos_theta + yOff * sin_theta + x0);
        int src_y = (int) (yOff * cos_theta - xOff * sin_theta + y0);
        if ((src_x >= 0) && (src_x < W) && (src_y >= 0) && (src_y < H)) {
          output_images[i * H * W + dest_y * W + dest_x] =
              input_images[i * H * W + src_y * W + src_x];
        } else {
          output_images[i * H * W + dest_y * W + dest_x] = 0.0f;
        }
      }
    }
  }
}

__global__ void rotate_image_kernel(float *input_images, float *output_images, int W, int H,
  float sin_theta, float cos_theta, int num_src_images) {
  float x0 = W / 2.0f;
  float y0 = H / 2.0f;

  int i = blockIdx.x / 4;
  int portion = blockIdx.x % 4;
  int dest_y = (portion * H / 4 + threadIdx.x);
  int dest_x = threadIdx.y;
  float xOff = dest_x - x0;
  float yOff = dest_y - y0;
  int src_x = (int) (xOff * cos_theta + yOff * sin_theta + x0);
  int src_y = (int) (yOff * cos_theta - xOff * sin_theta + y0);
  if ((src_x >= 0) && (src_x < W) && (src_y >= 0) && (src_y < H)) {
    output_images[i * H * W + dest_y * W + dest_x] =
    input_images[i * H * W + src_y * W + src_x];
  } else {
    output_images[i * H * W + dest_y * W + dest_x] = 0.0f;
  }
}

__global__ void test() {
  printf("In test funtion\n");
}



void rotate_image(float *input_images, float *output_images, int W, int H,
                  float sin_theta, float cos_theta, int num_src_images) {

  // (TODO) Upload input images to GPU
  CHECK_CUDA(hipMemcpy(input_images_gpu, input_images, sizeof(float) * W * H * num_src_images, hipMemcpyHostToDevice));

  // (TODO) Launch kernel on GPU
  dim3 gridDim(num_src_images * 4);
  dim3 blockDim(H / 4, W);
  rotate_image_kernel<<<gridDim, blockDim>>>(input_images_gpu, output_images_gpu, W, H, sin_theta, cos_theta, num_src_images);
  CHECK_CUDA(hipGetLastError());

  // (TODO) Download output images from GPU
  CHECK_CUDA(hipMemcpy(output_images, output_images_gpu, sizeof(float) * W * H * num_src_images, hipMemcpyDeviceToHost));
  printf("output: %f", output_images[0]);

  // DO NOT REMOVE; NEED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void rotate_image_init(int image_width, int image_height, int num_src_images) {
  // (TODO) Allocate device memory
  CHECK_CUDA(hipMalloc(&input_images_gpu, sizeof(float) * image_height * image_width * num_src_images));
  CHECK_CUDA(hipMalloc(&output_images_gpu, sizeof(float) * image_height * image_width * num_src_images));

  // DO NOT REMOVE; NEED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void rotate_image_cleanup() {
  // (TODO) Free device memory
  CHECK_CUDA(hipFree(input_images_gpu));
  CHECK_CUDA(hipFree(output_images_gpu));

  // DO NOT REMOVE; NEED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}
