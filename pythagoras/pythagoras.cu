
#include <hip/hip_runtime.h>
#include <cstdio>

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

__global__ void pythagoras(int *pa, int *pb, int *pc, int *presult) {
  int a = *pa;
  int b = *pb;
  int c = *pc;

  if ((a * a + b * b) == c * c)
    *presult = 1;
  else
    *presult = 0;
}

int main(int argc, char *argv[]) {
  if (argc != 4) {
    printf("Usage: %s <num 1> <num 2> <num 3>\n", argv[0]);
    return 0;
  }

  int a = atoi(argv[1]);
  int b = atoi(argv[2]);
  int c = atoi(argv[3]);
  int result = 0;

  // TODO: 1. allocate device memory
  int *ptr;
  hipMalloc(&ptr, sizeof(int) * 4);

  // TODO: 2. copy data to device
  hipMemcpy(ptr, &a, 4, hipMemcpyHostToDevice);
  hipMemcpy(ptr + 1, &b, 4, hipMemcpyHostToDevice);
  hipMemcpy(ptr + 2, &c, 4, hipMemcpyHostToDevice);

  // TODO: 3. launch kernel
  pythagoras<<<1,1>>>(ptr, ptr + 1, ptr + 2, ptr + 3);
  CHECK_CUDA(hipGetLastError());

  // TODO: 4. copy result back to host
  hipMemcpy(&result, ptr + 3, 4, hipMemcpyDeviceToHost);

  if (result) printf("YES\n");
  else printf("NO\n");

  return 0;
}
