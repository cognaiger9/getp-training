#include "hip/hip_runtime.h"
#include "matmul.h"
#include <cstdio>
#include <mma.h>

using namespace nvcuda;

#define CHECK_CUDA(call)                                                       \
  do {                                                                         \
    hipError_t status_ = call;                                                \
    if (status_ != hipSuccess) {                                              \
      fprintf(stderr, "CUDA error (%s:%d): %s\n", __FILE__, __LINE__,          \
              hipGetErrorString(status_));                                    \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  } while (0)

#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 16
#define WARP_SIZE 32
#define NUM_WARP ((WMMA_M * WMMA_N) / (WARP_SIZE))
#define C_LAYOUT wmma::mem_row_major

static __global__ void matmul_kernel(half *A, half *B, float *C, int M, int N,
                                     int K) {
  int gj = blockIdx.x;
  int gi = blockIdx.y;
  if (gi * WARP_SIZE >= M || gj * WARP_SIZE >= N)
    return; // boundary check
  int lj = threadIdx.x;
  int li = threadIdx.y;
  int warpId = li;

  __shared__ half Alocal[WARP_SIZE * WARP_SIZE];
  __shared__ half Blocal[WARP_SIZE * WARP_SIZE];

  // Declare the fragments
  wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major>
      a_frag;
  wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major>
      b_frag;
  wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;
  wmma::fill_fragment(c_frag, 0.0f);

  int A_row_index = (gi * WARP_SIZE + li);
  int B_col_index = (gj * WARP_SIZE + lj);

  for (int bk = 0; bk < K; bk += WARP_SIZE) {

    for (int offset = 0; offset < NUM_WARP; ++offset) {
      int A_col_index = bk + lj;
      Alocal[(li + offset * blockDim.y) * WARP_SIZE + lj] =
          ((A_row_index + offset * blockDim.y) < M && A_col_index < K)
              ? A[(A_row_index + offset * blockDim.y) * K + A_col_index]
              : (half)(0.0);

      int B_row_index = bk + li + (offset * blockDim.y);
      Blocal[(li + offset * blockDim.y) * WARP_SIZE + lj] =
          (B_row_index < K && B_col_index < N)
              ? B[B_row_index * N + B_col_index]
              : (half)(0.0);
    }
    __syncthreads();

    for (int i = 0; i < WARP_SIZE; i += WMMA_K) {
      int aCol = i;
      int aRow = (warpId / 2) * WMMA_M;
      int bCol = (warpId % 2) * WMMA_N;
      int bRow = i;

      wmma::load_matrix_sync(a_frag, Alocal + aCol + aRow * WARP_SIZE,
                             WARP_SIZE);
      wmma::load_matrix_sync(b_frag, Blocal + bCol + bRow * WARP_SIZE,
                             WARP_SIZE);

      wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
    }

    __syncthreads();
  }

  int cRow = (warpId / 2) * WMMA_M + blockIdx.y * blockDim.y * NUM_WARP;
  int cCol = (warpId % 2) * WMMA_N + blockIdx.x * blockDim.x;

  if (cRow + WMMA_M <= M && cCol + WMMA_N <= N) {
    wmma::store_matrix_sync(C + cCol + cRow * N, c_frag, N, C_LAYOUT);
  }
}

static half *A_gpu, *B_gpu;
static float *C_gpu;

void matmul(half *_A, half *_B, float *_C, int M, int N, int K) {
  CHECK_CUDA(
      hipMemcpy(A_gpu, _A, M * K * sizeof(half), hipMemcpyHostToDevice));
  CHECK_CUDA(
      hipMemcpy(B_gpu, _B, K * N * sizeof(half), hipMemcpyHostToDevice));
  dim3 blockDim(WARP_SIZE, 4);
  dim3 gridDim((N + WARP_SIZE - 1) / WARP_SIZE,
               (M + WARP_SIZE - 1) / WARP_SIZE);
  matmul_kernel<<<gridDim, blockDim>>>(A_gpu, B_gpu, C_gpu, M, N, K);
  CHECK_CUDA(hipGetLastError());
  CHECK_CUDA(
      hipMemcpy(_C, C_gpu, M * N * sizeof(float), hipMemcpyDeviceToHost));
}

void matmul_init(int M, int N, int K) {
  CHECK_CUDA(hipMalloc(&A_gpu, M * K * sizeof(half)));
  CHECK_CUDA(hipMalloc(&B_gpu, K * N * sizeof(half)));
  CHECK_CUDA(hipMalloc(&C_gpu, M * N * sizeof(float)));
}

void matmul_cleanup(half *_A, half *_B, float *_C, int M, int N, int K) {
  CHECK_CUDA(hipFree(A_gpu));
  CHECK_CUDA(hipFree(B_gpu));
  CHECK_CUDA(hipFree(C_gpu));
}
