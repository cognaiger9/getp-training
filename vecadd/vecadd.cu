#include "hip/hip_runtime.h"
#include <cstdio>

#include "vecadd.h"

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

__global__ void vecadd_kernel(const int N, const float *a, const float *b, float *c) {
  int tidx = blockIdx.x * blockDim.x + threadIdx.x;
  if (tidx < N) {
    c[tidx] = a[tidx] + b[tidx];
  }
}

// Device(GPU) pointers
static float *A_gpu, *B_gpu, *C_gpu;
#define BLOCKSIZE 1024

void vecadd(float *_A, float *_B, float *_C, int N) {
  // (TODO) Upload A and B vector to GPU
  vecadd_init(N);
  hipMemcpy(A_gpu, _A, sizeof(float) * N, hipMemcpyHostToDevice);
  hipMemcpy(B_gpu, _B, sizeof(float) * N, hipMemcpyHostToDevice);

  // Launch kernel on a GPU
  dim3 blockDim(BLOCKSIZE);
  dim3 gridDim((N + blockDim.x - 1) / blockDim.x);
  vecadd_kernel<<<gridDim, blockDim>>>(N, A_gpu, B_gpu, C_gpu);

  // (TODO) Download C vector from GPU
  hipMemcpy(_C, C_gpu, sizeof(float) * N, hipMemcpyDeviceToHost);

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void vecadd_init(int N) {
  // (TODO) Allocate device memory
  hipMalloc(&A_gpu, N * sizeof(float));
  hipMalloc(&B_gpu, N * sizeof(float));
  hipMalloc(&C_gpu, N * sizeof(float));

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void vecadd_cleanup(float *_A, float *_B, float *_C, int N) {
  // (TODO) Do any post-vecadd cleanup work here.
  hipFree(A_gpu);
  hipFree(B_gpu);
  hipFree(C_gpu);

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}
