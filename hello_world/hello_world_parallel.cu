
#include <hip/hip_runtime.h>
#include <cstdio>

// called from CPU and executed in GPU
__global__ void hello_world() {
  int tidx = threadIdx.x + blockIdx.x * blockDim.x;
  printf("Device(GPU) Thread %d: Hello, World!\n", tidx);
}

int main()
{
  hello_world<<<4, 4>>>();
  hipDeviceSynchronize();
  return 0;
}
